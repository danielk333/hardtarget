#include "hip/hip_runtime.h"
// header file for the plasmaline project
#include "gmfgpu.h"

void print_devices()
{
  int nDevices;

  int ret=hipGetDeviceCount(&nDevices);
 
  for (int i = 0; i < nDevices; i++)
  {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}


/*
  For each range gate (i), multiply transmit pulse with range delayed echo
  each range gate has a nfft2 length block in d_z_echo.
  all of these will be fft'ed in parallel with CuFFT at the next stage.

 */
__global__ void form_input(hipfftComplex *z_tx, int z_tx_len, int nfft2,
                           hipfftComplex *z_rx, int *rgs, int dec,
			   hipfftComplex *d_z_echo, int *tx_idx, int nzi)
{
  int i = blockIdx.x;
  int rg=rgs[i];
  for(int ni=0 ; ni<nzi; ni++)
  {
    int ti=tx_idx[ni];
    d_z_echo[i*nfft2+ti/dec] = hipCaddf(d_z_echo[i*nfft2+ti/dec],hipCmulf(z_tx[ti], z_rx[rg+ti]));
  }
}

/*
  For each range gate (i), find the value of the spectrum that has highest power.
  Also store DC value for noise floor determination.
 */
__global__ void peak_find(hipfftComplex *z_out, float *gmf_vec, float *gmf_dc_vec, float *v_vec, float *a_vec, int nfft2, int acc_idx)
{
  int i = blockIdx.x;
  if(acc_idx==0)
    gmf_dc_vec[i]=z_out[i*nfft2].x*z_out[i*nfft2].x+z_out[i*nfft2].y*z_out[i*nfft2].y;
  for(int j=0;j<nfft2;j++)
  {
    float this_gmf=z_out[i*nfft2+j].x*z_out[i*nfft2+j].x+z_out[i*nfft2+j].y*z_out[i*nfft2+j].y;
    if(this_gmf>gmf_vec[i])
    {
      gmf_vec[i]=this_gmf;
      v_vec[i]=(float)j;
      a_vec[i]=(float)acc_idx ;
    }
  }
}

/*
  Multiply echo with acceleration phasor, store input in d_z_in
 */
__global__ void phasor_multiply(hipfftComplex *d_z_echo, hipfftComplex *d_z_in, int nfft2, int i, hipfftComplex *d_acc_phasors)
{
  int rgi = blockIdx.x;
  /* 
     tbd: only multiply non-zero values.
  */
  for(int j=0;j<nfft2;j++)
  {
    d_z_in[rgi*nfft2+j]=hipCmulf(d_z_echo[rgi*nfft2+j],d_acc_phasors[i*nfft2+j]);
  }
}

/* 
   This is the main code. If you have N GPUs, you can run N gmf functions in parallel.
*/
extern "C" int gmf(float *z_tx, int z_tx_len, float *z_rx, int z_rx_len, float *acc_phasors, int n_accs,
       	       	   float *rgs, int n_rg, int dec, float *gmf_vec, float *gmf_dc_vec, float *v_vec, float *a_vec, int rank)
{
  hipSetDevice(rank);
  // initializing pointers to device (GPU) memory, denoted with "d_"
  hipfftComplex *d_z_tx;
  hipfftComplex *d_z_rx;
  //  hipfftComplex *d_z_out;
  hipfftComplex *d_z_echo;  
  hipfftComplex *d_z_in;      
  hipfftComplex *d_acc_phasors;
  float *d_gmf_vec;
  float *d_gmf_dc_vec;
  float *d_v_vec;
  float *d_a_vec;
  int *d_tx_idx;
  
  int *d_rgs;
  int *h_rgs;
  h_rgs=(int *)malloc(sizeof(int)*n_rg);
  for(int i=0;i<n_rg;i++)
  {
    h_rgs[i]=(int)rgs[i];
  }
      
  float *tx_power;
  tx_power=(float*)malloc(sizeof(float)*z_tx_len);
  
  int n_nonzero_tx=0;
  for(int ti=0 ; ti<z_tx_len ; ti++)
  {
    tx_power[ti]=z_tx[2*ti]*z_tx[2*ti] + z_tx[2*ti+1]*z_tx[2*ti+1];
    if(tx_power[ti] > 1e-10)
      n_nonzero_tx++;
  }

  int *tx_idx;
  tx_idx=(int*)malloc(sizeof(int)*n_nonzero_tx);
  int nzi=0;
  for(int ti=0 ; ti<z_tx_len ; ti++)
  {
    if(tx_power[ti] > 1e-10)
    {
      tx_idx[nzi]=ti;
      nzi++;
    }
  }
  
  int nfft2;
  
  nfft2=(int)(z_tx_len/dec);    

  // allocating device memory to the above pointers
  // the signal and echo here are only one row of the CPU data (one time step)
  int res= hipMalloc((void **) &d_z_tx, sizeof(hipfftComplex) * z_tx_len);
  if (res != hipSuccess)
  {
    printf("error %d\n",res);
    fprintf(stderr, "Cuda error: Failed to allocate tx\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **) &d_rgs, sizeof(int) * n_rg) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate tx\n");
    exit(EXIT_FAILURE);
  }
  
  if (hipMalloc((void **) &d_tx_idx, sizeof(int) * nzi) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate tx\n");
    exit(EXIT_FAILURE);
  }


  if (hipMalloc((void **) &d_z_rx, sizeof(hipfftComplex) * z_rx_len) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate echo\n");
    exit(EXIT_FAILURE);
  }
    
  if (hipMalloc((void **) &d_z_in, sizeof(hipfftComplex) * nfft2 * n_rg)
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate batch\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **) &d_z_echo, sizeof(hipfftComplex) * nfft2 * n_rg)
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate batch\n");
    exit(EXIT_FAILURE);
  }
  //  if (hipMalloc((void **) &d_z_out, sizeof(hipfftComplex) * nfft2 * n_rg)
  //  != hipSuccess)
  //  {
  // fprintf(stderr, "Cuda error: Failed to allocate batch\n");
  // exit(EXIT_FAILURE);
  // }
  if (hipMalloc((void **) &d_acc_phasors, sizeof(hipfftComplex) * n_accs*nfft2)
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **) &d_gmf_vec, sizeof(float) * n_rg)
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **) &d_gmf_dc_vec, sizeof(float) * n_rg)
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **) &d_v_vec, sizeof(float) * n_rg)
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **) &d_a_vec, sizeof(float) * n_rg)
      != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to allocate spectrum\n");
    exit(EXIT_FAILURE);
  }
  //  printf("malloced stuff\n");
  // initializing 1D FFT plan, this will tell cufft execution how to operate
  // cufft is well optimized and will run with different parameters than above
  hipfftHandle plan;
  if (hipfftPlan1d(&plan, nfft2, HIPFFT_C2C, n_rg) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Plan creation failed\n");
    exit(EXIT_FAILURE);
  }
  //  printf("planned fft\n");
  // execution of the prepared kernels n_ipp times
  // ensure empty device spectrum
  if (hipMemset(d_z_in, 0, sizeof(hipfftComplex) * nfft2*n_rg) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipMemset(d_z_echo, 0, sizeof(hipfftComplex) * nfft2*n_rg) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
    exit(EXIT_FAILURE);
  }

  // copying n_ipp'th row of host data into device
  if (hipMemcpy(d_z_tx, z_tx, sizeof(hipfftComplex) * z_tx_len,
		 hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, tx HtD\n");
    exit(EXIT_FAILURE);
  }
  // copying n_ipp'th row of host data into device
  if (hipMemcpy(d_rgs, h_rgs, sizeof(int) * n_rg,
		 hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, tx HtD\n");
    exit(EXIT_FAILURE);
  }

  // copying n_ipp'th row of host data into device
  if (hipMemcpy(d_tx_idx, tx_idx, sizeof(int) * nzi,
		 hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, tx HtD\n");
    exit(EXIT_FAILURE);
  }
  

  // copying n_ipp'th row of host data into device
  if (hipMemcpy(d_acc_phasors, acc_phasors, sizeof(hipfftComplex) * nfft2*n_accs,
		 hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, tx HtD\n");
    exit(EXIT_FAILURE);
  }

  if (hipMemcpy(d_z_rx, z_rx, sizeof(hipfftComplex) * z_rx_len,
		 hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, echo HtD\n");
    exit(EXIT_FAILURE);
  }

  //  if (hipMalloc((void **) &d_acc_phasors, sizeof(hipfftComplex) * n_accs*nfft2)
  
        
  // form input
  form_input<<<n_rg,1>>>(d_z_tx, z_tx_len, nfft2, d_z_rx, d_rgs, dec, d_z_echo, d_tx_idx, nzi);
  
  for(int i=0 ; i < n_accs; i++)
  {
    if (hipMemset(d_z_in, 0, sizeof(hipfftComplex) * nfft2*n_rg) != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to zero device spectrum\n");
      exit(EXIT_FAILURE);
    }
    phasor_multiply<<<n_rg,1>>>(d_z_echo, d_z_in, nfft2, i, d_acc_phasors);
    
    // cufft kernel execution
    if (hipfftExecC2C(plan, (hipfftComplex *)d_z_in, (hipfftComplex *)d_z_in, HIPFFT_FORWARD)
	       != HIPFFT_SUCCESS)
    {
      fprintf(stderr, "CUFFT error: ExecC2C Forward failed\n");
      exit(EXIT_FAILURE);
    }
    peak_find<<<n_rg,1>>>(d_z_in, d_gmf_vec, d_gmf_dc_vec, d_v_vec, d_a_vec, nfft2, i);
  }


  // copying device resultant spectrum to host, now able to be manipulated
  if (hipMemcpy(gmf_vec, d_gmf_vec, sizeof(float) * n_rg,
		 hipMemcpyDeviceToHost) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
    exit(EXIT_FAILURE);
  }
  // copying device resultant spectrum to host, now able to be manipulated
  if (hipMemcpy(gmf_dc_vec, d_gmf_dc_vec, sizeof(float) * n_rg,
		 hipMemcpyDeviceToHost) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
    exit(EXIT_FAILURE);
  }
  // copying device resultant spectrum to host, now able to be manipulated
  if (hipMemcpy(v_vec, d_v_vec, sizeof(float) * n_rg,
		 hipMemcpyDeviceToHost) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
    exit(EXIT_FAILURE);
  }
  // copying device resultant spectrum to host, now able to be manipulated
  if (hipMemcpy(a_vec, d_a_vec, sizeof(float) * n_rg,
		 hipMemcpyDeviceToHost) != hipSuccess)
  {
    fprintf(stderr, "Cuda error: Memory copy failed, spectrum DtH\n");
    exit(EXIT_FAILURE);
  }

  free(h_rgs);
  free(tx_idx);
  free(tx_power);
  
  // memory clean up
  if (hipFree(d_z_tx) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free tx\n");
    exit(EXIT_FAILURE);
  }

  if (hipFree(d_rgs) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free tx\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_z_rx) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free echo\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_z_in) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free batch\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_z_echo) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free batch\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_v_vec) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_a_vec) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_gmf_vec) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_gmf_dc_vec) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_acc_phasors) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipFree(d_tx_idx) != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to free spectrum\n");
    exit(EXIT_FAILURE);
  }
  if (hipfftDestroy(plan) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Failed to destroy plan\n");
    exit(EXIT_FAILURE);
  }
    return(1);
}
